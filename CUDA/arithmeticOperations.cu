#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void arth(int a, int b, int *c, int *d, int *e, int *f)
{
  *c = a + b;
  *d = a -b;
  *e = a * b;
  *f = a / b;
}

int main()
{
  int a,b,c,d,e,f;
  int *dev_c,*dev_d,*dev_e,*dev_f;
  
  scanf("%d%d",&a,&b);
  
  hipMalloc((void**)&dev_c, sizeof(int));
  hipMalloc((void**)&dev_d, sizeof(int));
  hipMalloc((void**)&dev_e, sizeof(int));
  hipMalloc((void**)&dev_f, sizeof(int));
  
  arth<<<1,1>>>(a,b,dev_c,dev_d,dev_e,dev_f);
  
  hipMemcpy(&c, dev_c, sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(&d, dev_d, sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(&e, dev_e, sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(&f, dev_f, sizeof(int), hipMemcpyDeviceToHost);
  
  printf("%d+ %dis %d\n", a, b, c);
  printf("%d-%dis %d\n", a, b, d);
  printf("%d* %dis %d\n", a, b, e);
  printf("%d/ %dis %d\n", a, b, f);
  
  hipFree(dev_c);
  hipFree(dev_d);
  hipFree(dev_e);
  hipFree(dev_f);
  
  return 0;
}
