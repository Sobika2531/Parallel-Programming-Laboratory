#include <stdio.h>
#include <hip/hip_runtime.h>

#define n 10

__global__ void add(int*a, int*max)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  
  if(i < n)
  {
    for(i=0;i<n;i++)
    {
      if(a[i]>*max)
        *max=a[i];
    }
  }
}

int main()
{
  int a[n];
  int i;
  int max;
  int* dev_a;
  int* dev_max;
  
  hipMalloc((void**)&dev_max, sizeof(int));
  hipMalloc((void**)&dev_a, n * sizeof(int));
  
  printf("\narray elements:\n");
  
  for(i=0;i<n;i++)
  {
    scanf("%d",&a[i]);
  }
  
  max = a[0];
  
  hipMemcpy(dev_a, a, n * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(dev_max, &max, sizeof(int), hipMemcpyHostToDevice);
  add<<<1,1>>>(dev_a, dev_max);
  hipMemcpy(&max, dev_max, sizeof(int), hipMemcpyDeviceToHost);
  
  printf("\nMax is %d\n",max);
  
  hipFree(dev_max);
  hipFree(dev_a);
  hipDeviceReset();
  
  return 0;
}
