#include <stdio.h>
#include <hip/hip_runtime.h>

#define n 5

__global__ void add(int *a, int *b, int *c)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  
  if(i < n) 
  {
    c[i] = a[i] + b[i];
  }
}

int main()
{
  int a[n];
  int b[n];
  int i;
  int c[n]= {0};
  int* dev_a;
  int* dev_b;
  int* dev_c;
  
  hipMalloc((void**)&dev_c, n * sizeof(int));
  hipMalloc((void**)&dev_a, n * sizeof(int));
  hipMalloc((void**)&dev_b, n * sizeof(int));
  
  printf("\narray elements (1st):\n");
  
  for(i=0;i<n;i++)
  {
    scanf("%d",&a[i]);
  }
  
  printf("\narray elements (2nd):\n");
  
  for(i=0;i<n;i++)
  {
    scanf("%d",&b[i]);
  }
  
  hipMemcpy(dev_a, a, n * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(dev_b, b, n* sizeof(int), hipMemcpyHostToDevice);
  
  add<<<n,1>>>(dev_a, dev_b, dev_c);
  
  hipMemcpy(c, dev_c, n* sizeof(int), hipMemcpyDeviceToHost);
  
  printf("\nsum is\n");
  
  for(i=0;i<n;i++)
  {
    printf("%d\n",c[i]);
  }
  
  hipFree(dev_c);
  hipFree(dev_a);
  hipFree(dev_b);
  hipDeviceReset();
  
  return 0;
}
